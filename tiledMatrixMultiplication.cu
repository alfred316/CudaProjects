#include "hip/hip_runtime.h"
//Alfred Shaker
//November 13th 2015
//Tiled matrix multiplication

#include <stdlib.h>
#include <stdio.h>

//tile dimention
#define TILE_DIM 32 

//kernel function
__global__ void tileMatMul(float* matA, float* matB, float* matC, int aRows, int aCols, 
				int bRows, int bCols, int cRows, int cCols)
{
	//define row and column values
	int Row = blockIdx.y * TILE_DIM + threadIdx.y;
	int Col = blockIdx.x * TILE_DIM + threadIdx.x;

	//shared memory arrays
	__shared__ float sharedMatA[TILE_DIM][TILE_DIM];
	__shared__ float sharedMatB[TILE_DIM][TILE_DIM];

	float cResultValue = 0.0;

	//calculate tiled matrix multiplication on shared memory
	for(int i = 0; i < (aCols-1)/TILE_DIM+1; ++i)
	{
	    if(Row < aRows && i*TILE_DIM+threadIdx.x < aCols)
	    {
	        sharedMatA[threadIdx.y][threadIdx.x] = matA[Row*aCols + i*TILE_DIM+threadIdx.x];
	    }
	    else
	        sharedMatA[threadIdx.y][threadIdx.x] = 0.0;

	    if(Col < bCols && i*TILE_DIM+threadIdx.y < cRows)
	        sharedMatB[threadIdx.y][threadIdx.x] = matB[(i*TILE_DIM+threadIdx.y)*bCols+Col];
	    else
	        sharedMatB[threadIdx.y][threadIdx.x] = 0.0;

	    __syncthreads();

	    for(int j = 0; j < TILE_DIM; ++j)
	        cResultValue += sharedMatA[threadIdx.y][j] * sharedMatB[j][threadIdx.x];
	 
	   __syncthreads();
	}

	//put the results in the result matrix
	if(Row < cRows && Col < cCols)
	    matC[Row*cCols+Col] = cResultValue;

}


int main()
{
	//define the host matrices
	float *hMatA, *hMatB, *hMatC;
	//define device matrices
	float *dMatA, *dMatB, *dMatC;

	//define matrix dimentions
	int aRows = 512;
	int aCols = 512; 
	int bRows = 512;
	int bCols = 512; 
	int cRows, cCols;
	
	//allocate space for host matrices
	hMatA = (float *) malloc(sizeof(float)*aRows*aCols);
	hMatB = (float *) malloc(sizeof(float)*bRows*bCols);
	
	//fill up the matrices with reamdom float values
	//between 0.0 and 1.0
	for(int i = 0; i < aRows*aCols; ++i)
	{
	    hMatA[i] = (float)rand()/(float)(RAND_MAX/1.0);
	    hMatB[i] = (float)rand()/(float)(RAND_MAX/1.0);
	}	

	//define the dimentions for the result variable	
	cRows = aRows;
	cCols = bCols;

	//allocate host result matrix
	hMatC = (float *) malloc(sizeof(float)*cRows*cCols);
	
	//cuda alloate the device  matrices 
	hipMalloc((void**)&dMatA, sizeof(float)*aRows*aCols);
	hipMalloc((void**)&dMatB, sizeof(float)*bRows*bCols);
	hipMalloc((void**)&dMatC, sizeof(float)*cRows*cCols);

	//copy data from host to device matrices
	hipMemcpy(dMatA, hMatA, sizeof(float)*aRows*aCols, hipMemcpyHostToDevice);
	hipMemcpy(dMatB, hMatB, sizeof(float)*bRows*bCols, hipMemcpyHostToDevice);

	//define grid and block dimentions
	dim3 dimGrid((cCols - 1)/TILE_DIM+1, (cRows - 1)/TILE_DIM+1, 1);
	dim3 dimBlock(TILE_DIM, TILE_DIM, 1);
	
	//call kernel function
	tileMatMul<<<dimGrid,dimBlock>>>(dMatA, dMatB, dMatC, aRows, aCols, bRows, bCols, cRows, cCols);
	
	//sync the threads
	hipDeviceSynchronize();
	
	//copy result from device to host
	hipMemcpy(hMatC, dMatC, sizeof(float)*cRows*cCols, hipMemcpyDeviceToHost);
	
	//print first 100 results
	for(int q = 0; q < 100; ++q)
	{
	    printf("Result matrix #%d: %f\n",q, hMatC[q]);
	}

	//free device variables
	hipFree(dMatA);
	hipFree(dMatB);
	hipFree(dMatC);

	//free host variables
	free(hMatA);
	free(hMatB);
	free(hMatC);

	return 0;	
		
}




